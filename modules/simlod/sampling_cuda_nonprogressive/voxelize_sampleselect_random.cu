#include "hip/hip_runtime.h"


namespace sampleselect_random{

#include <hip/hip_cooperative_groups.h>
#include "lib.h.cu"
#include "methods_common.h.cu"
#include <hiprand/hiprand_kernel.h>

namespace cg = cooperative_groups;

void voxelizePrimitives(Node* node, vec3 boxSize, uint32_t num, 
	Point* points, uint32_t gridSize, float fGridSize, 
	uint64_t* voxelGrid, uint32_t* voxelBufferSize, Point* voxelBuffer,
	int childIndex, hiprandStateXORWOW_t* randStates, hiprandStateXORWOW_t* thread_random_state
){

	processRange(0, num, [&](int pointIndex){

		Point point = points[pointIndex];

		float fx = fGridSize * (point.x - node->min.x) / boxSize.x;
		float fy = fGridSize * (point.y - node->min.y) / boxSize.y;
		float fz = fGridSize * (point.z - node->min.z) / boxSize.z;

		uint32_t ix = clamp(fx, 0.0f, fGridSize - 1.0f);
		uint32_t iy = clamp(fy, 0.0f, fGridSize - 1.0f);
		uint32_t iz = clamp(fz, 0.0f, fGridSize - 1.0f);

		uint32_t voxelIndex = ix + gridSize * iy + gridSize * gridSize * iz;

		// uint64_t randomNumber = static_cast<uint64_t>(hiprand (&randStates[pointIndex % 1024]));
		uint64_t randomNumber = hiprand(thread_random_state);
		// uint64_t randomNumber = pointIndex;
		uint64_t encoded = (randomNumber << 32ull) | (static_cast<uint64_t>(childIndex) << 24ull) | (static_cast<uint64_t>(pointIndex));
		auto old = atomicMax(&voxelGrid[voxelIndex], encoded);

		if(old == 0){
			uint32_t rindex = atomicAdd(voxelBufferSize, 1);
			voxelBuffer[rindex].color = voxelIndex;
		}


		// { 
		// 	auto warp = cg::coalesced_threads();
		// 	auto group = cg::labeled_partition(warp, voxelIndex);

		// 	if(group.thread_rank() == 0){
		// 		auto old = atomicCAS(&voxelGrid[voxelIndex], 0, (uint64_t)&points[pointIndex]);

		// 		if(old == 0){
		// 			uint32_t rindex = atomicAdd(voxelBufferSize, 1);
		// 			voxelBuffer[rindex].color = voxelIndex;
		// 		}
		// 	}
		// }
		
	});
}

void main_voxelize(
	Allocator& allocator, 
	Box3 box, 
	int numPoints,
	void* nnnodes,
	uint32_t numNodes,
	void* sssorted)
{
	// Point* sorted = (Point*)sssorted;
	Node* nodes = (Node*)nnnodes;

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	NodePtr* workload              =  allocator.alloc<NodePtr*>(sizeof(Node) * numNodes, "workload");
	uint32_t& workloadSize         = *allocator.alloc<uint32_t*>(sizeof(uint32_t), "workload counter");
	uint32_t& totalVoxelBufferSize = *allocator.alloc<uint32_t*>(4, "total voxel buffer counter");
	uint32_t& nodeVoxelBufferSize  = *allocator.alloc<uint32_t*>(4, "node voxel buffer counter");
	uint32_t& clearCounter         = *allocator.alloc<uint32_t*>(4, "clear counter");

	if(isFirstThread()){
		totalVoxelBufferSize = 0;
		nodeVoxelBufferSize = 0;
		clearCounter = 0;
	}

	grid.sync();

	{
		int gridSize        = VOXEL_GRID_SIZE;
		float fGridSize     = gridSize;
		int numCells        = gridSize * gridSize * gridSize;
		uint64_t* voxelGrid = allocator.alloc<uint64_t*>(sizeof(uint64_t) * numCells, "voxel sampling grid RG");

		hiprandStateXORWOW_t* randStates = allocator.alloc<hiprandStateXORWOW_t*>(sizeof(hiprandStateXORWOW_t) * 1024, "CUDA random states");
		// Initialize the CUDA random states once (hardcoded seed)
		processRange(0, 1024, [&](int sequence) {
			hiprand_init (18324725504264168865u, sequence, 0, &randStates[sequence]);
		});

		hiprandStateXORWOW_t thread_random_state;
		hiprand_init(grid.thread_rank(), 0, 0, &thread_random_state);

		clearBuffer(voxelGrid, 0, sizeof(uint64_t) * numCells, 0);

		for(int abc = 0; abc < 12; abc++){

			grid.sync();

			auto t_create_workload = Timer::start("create workload");
			{ // FIND DEEPEST EMPTY INNER NODES (TO FILL WITH VOXELS)
				if(isFirstThread()){ 
					workloadSize = 0;
				}
				grid.sync();

				processRange(0, numNodes, [&](int nodeIndex){
					Node* node = &nodes[nodeIndex];
					
					int numSamplesInChildren = 0;
					bool allChildrenNonempty = true;
					for(int childIndex = 0; childIndex < 8; childIndex++){
						Node* child = node->children[childIndex];

						if(child){
							numSamplesInChildren += child->numPoints + child->numVoxels;
							if((child->numPoints + child->numVoxels) == 0){
								allChildrenNonempty = false;
							}
						}
					}
				
					bool isEmpty = node->numPoints == 0 && node->numVoxels == 0;

					if(isEmpty && allChildrenNonempty){
						uint32_t targetIndex = atomicAdd(&workloadSize, 1);
						workload[targetIndex] = node;
					}
				});

				grid.sync();
			}
			t_create_workload.stop();


			if(workloadSize == 0){
				// we're done!
				break;
			}

			{ // CREATE VOXEL LOD FOR DEEPEST EMPTY INNER NODES

				for(int nodeIndex = 0; nodeIndex < workloadSize; nodeIndex++){
					
					Node* node = workload[nodeIndex];
					Point* voxelBuffer = reinterpret_cast<Point*>(allocator.buffer + allocator.offset + sizeof(Point) * totalVoxelBufferSize);

					grid.sync();

					vec3 boxSize = node->max - node->min;

					grid.sync();

					// (Re-)Initialize the CUDA random states for each node
					/* processRange(0, 1024, [&](int sequence) {
						hiprand_init (nodeIndex, sequence, 0, &randStates[sequence]);
					}); */

					// auto t_voxelize = Timer::start("voxelize");
					for(int childIndex = 0; childIndex < 8; childIndex++){
						Node* child = node->children[childIndex];
						if(child == nullptr) continue;

						grid.sync();

						// VOXELIZE POINTS
						voxelizePrimitives(node, boxSize, child->numPoints, 
							child->points, gridSize, fGridSize, 
							voxelGrid, &nodeVoxelBufferSize, voxelBuffer, childIndex, 
							randStates, &thread_random_state);

						grid.sync();

						// VOXELIZE VOXELS
						voxelizePrimitives(node, boxSize, child->numVoxels, 
							child->voxels, gridSize, fGridSize,
							voxelGrid, &nodeVoxelBufferSize, voxelBuffer, childIndex, 
							randStates, &thread_random_state);
					}
					// t_voxelize.stop();

					grid.sync();

					// EXTRACT VOXELS
					// auto t_extract = Timer::start("extract");
					processRange(0, nodeVoxelBufferSize, [&](int rindex){
						
						uint32_t index = voxelBuffer[rindex].color;
						uint64_t encoded = voxelGrid[index];

						if(encoded != 0) {
							uint32_t pointIndex = static_cast<uint32_t>(encoded & 0x7FFFFF);
							uint32_t childIndex = static_cast<uint32_t>((encoded >> 24) & 0xFF);
							Node* child = node->children[childIndex];
							bool isPoint = child->numPoints > 0;
							Point* ptr = isPoint ? child->points + pointIndex : child->voxels + pointIndex;

							if(ptr != nullptr){
								Point point = *ptr;

								voxelBuffer[rindex] = point;
								voxelGrid[index] = 0;
							}
						}
					});
					// t_extract.stop();

					grid.sync();

					// { // very slightly more efficient 64 bit clear
					// 	uint64_t totalThreadCount = blockDim.x * gridDim.x;
					// 	uint64_t itemsPerThread = (numCells / totalThreadCount) + 1;
					// 	uint64_t itemsPerBlock = itemsPerThread * block.num_threads();
					// 	uint64_t blockFirst = blockIdx.x * itemsPerBlock;

					// 	uint64_t* buffer = reinterpret_cast<uint64_t*>(voxelGrid);

					// 	for(int i = 0; i < itemsPerThread; i++){
					// 		uint32_t index = blockFirst + i * block.num_threads() + threadIdx.x;

					// 		if(index >= 2 * numCells) continue;

					// 		buffer[index] = 0;
					// 	}
					// }


					grid.sync();
					
					if(isFirstThread()){
						node->voxels = voxelBuffer;
						node->numVoxels = nodeVoxelBufferSize;

						atomicAdd(&totalVoxelBufferSize, nodeVoxelBufferSize);
						atomicExch(&nodeVoxelBufferSize, 0);
					}
					
					grid.sync();
				}
			}
		}

		grid.sync();

		// post-allocation of generated voxels
		// we dynamically placed voxels starting from previous allocator.offset 
		// now that we've done that, we know how much memory we needed and update the allocator
		allocator.alloc<Point*>(sizeof(Point) * totalVoxelBufferSize, "all voxels");
	}

	grid.sync();

	if(false)
	if(isFirstThread())
	{ // DEBUG

		printf("#voxels:                 %8i \n", totalVoxelBufferSize);
		printf("allocated memory:        "); //%i MB \n", allocator.offset / (1024 * 1024));
		printNumber(allocator.offset, 13);
		printf("\n");
		printf("clear counter: %i \n", clearCounter);
	}

}

};