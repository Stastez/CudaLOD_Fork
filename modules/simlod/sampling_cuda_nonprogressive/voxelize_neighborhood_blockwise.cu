#include "hip/hip_runtime.h"


namespace voxelize_neighborhood_blockwise{

#include <hip/hip_cooperative_groups.h>
#include "lib.h.cu"
#include "methods_common.h.cu"

namespace cg = cooperative_groups;

constexpr int clearGridSize = 8;

void computeWorkload(Node* nodes, uint32_t numNodes, NodePtr* workload, uint32_t& workloadSize){

	auto grid = cg::this_grid();

	if(isFirstThread()){ 
		workloadSize = 0;
	}
	grid.sync();

	processRange(0, numNodes, [&](int nodeIndex){
		Node* node = &nodes[nodeIndex];
		
		int numSamplesInChildren = 0;
		bool allChildrenNonempty = true;
		for(int childIndex = 0; childIndex < 8; childIndex++){
			Node* child = node->children[childIndex];

			if(child){
				numSamplesInChildren += child->numPoints + child->numVoxels;
				if((child->numPoints + child->numVoxels) == 0){
					allChildrenNonempty = false;
				}
			}
		}
	
		bool isEmpty = node->numPoints == 0 && node->numVoxels == 0;

		if(isEmpty && allChildrenNonempty){
			uint32_t targetIndex = atomicAdd(&workloadSize, 1);
			workload[targetIndex] = node;
		}
	});

	grid.sync();

}

// constexpr int MODE_CENTRAL = 0;
// constexpr int MODE_ADJACENT = 1;

void voxelizePrimitives_central(
	Point* points, 
	uint32_t numPoints, 
	Node* node, 
	int gridSize,
	uint32_t* voxelGrid,
	vec3 boxSize, 
	uint32_t& sh_numAccepted, 
	uint32_t* accepted,
	uint32_t* sh_clearGrid
){
	auto block = cg::this_thread_block();

	float fGridSize = gridSize;

	int numIterations = numPoints / block.num_threads() + 1;
	for(int it = 0; it < numIterations; it++){
		int pointIndex = block.num_threads() * it + block.thread_rank();

		if(pointIndex >= numPoints) continue;

		Point point = points[pointIndex];

		// project to node's 128³ sample grid
		float fx = fGridSize * (point.x - node->min.x) / boxSize.x;
		float fy = fGridSize * (point.y - node->min.y) / boxSize.y;
		float fz = fGridSize * (point.z - node->min.z) / boxSize.z;

		vec3 pos = {fx, fy, fz};

		{

			vec3 samplePos = vec3(
				floor(fx) + 0.5f,
				floor(fy) + 0.5f,
				floor(fz) + 0.5f
			);

			float dx = (pos.x - samplePos.x);
			float dy = (pos.y - samplePos.y);
			float dz = (pos.z - samplePos.z);
			float ll = (dx * dx + dy * dy + dz * dz);
			float w = 0.0f;

			float l = sqrt(ll);

			if(ll < 1.0f){
				// exponential filter
				// w = __expf(-ll * 0.5f);
				// w = clamp(w, 0.0f, 1.0f);
				
				// linear filter
				w = 1.0 - l;
			}else{
				w = 0.0;
			}

			if(w > 0.0f){

				uint64_t W = clamp(100.0f * w, 1.0f, 100.0f);

				uint32_t ix = clamp(samplePos.x, 0.0f, fGridSize - 1.0f);
				uint32_t iy = clamp(samplePos.y, 0.0f, fGridSize - 1.0f);
				uint32_t iz = clamp(samplePos.z, 0.0f, fGridSize - 1.0f);

				uint32_t voxelIndex = ix + gridSize * iy + gridSize * gridSize * iz;


				// if(ox == 0.0f && oy == 0.0f && oz == 0.0f){
				// 	uint32_t res = atomicOr(&voxelGrid[4 * voxelIndex + 3], (1u << 31u));
				// 	bool isNewlyOccupied = (res & (1u << 31u)) == 0;
					
				// 	if (isNewlyOccupied){
				// 		uint32_t acceptedIndex = atomicAdd(&sh_numAccepted, 1);
				// 		accepted[acceptedIndex] = voxelIndex;
				// 	}
				// }

				uint64_t* cell = (uint64_t*)&voxelGrid[4 * voxelIndex + 0];

				uint8_t* rgba = (uint8_t*)&point.color;
				uint64_t R = W * rgba[0];
				uint64_t G = W * rgba[1];
				uint64_t B = W * rgba[2];
				atomicAdd(cell + 0, uint64_t(R | (G << 32)));
				uint64_t old = atomicAdd(cell + 1, uint64_t(B | (W << 32)));


				bool isNewlyOccupied = (old >> 32) == 0;
				if (isNewlyOccupied){
					uint32_t acceptedIndex = atomicAdd(&sh_numAccepted, 1);
					accepted[acceptedIndex] = voxelIndex;
				}
			}
		}
	}
}

void voxelizePrimitives_neighbors(
	Point* points, 
	uint32_t numPoints, 
	Node* node, 
	int gridSize,
	uint32_t* voxelGrid,
	vec3 boxSize, 
	uint32_t& sh_numAccepted, 
	uint32_t* accepted,
	uint32_t* sh_clearGrid
){
	auto block = cg::this_thread_block();

	float fGridSize = gridSize;

	int numIterations = numPoints / block.num_threads() + 1;
	for(int it = 0; it < numIterations; it++){
		int pointIndex = block.num_threads() * it + block.thread_rank();

		if(pointIndex >= numPoints) continue;

		Point point = points[pointIndex];

		// project to node's 128³ sample grid
		float fx = fGridSize * (point.x - node->min.x) / boxSize.x;
		float fy = fGridSize * (point.y - node->min.y) / boxSize.y;
		float fz = fGridSize * (point.z - node->min.z) / boxSize.z;

		vec3 pos = {fx, fy, fz};

		for(float oz : {-1.0f, 0.0f, 1.0f})
		for(float oy : {-1.0f, 0.0f, 1.0f})
		for(float ox : {-1.0f, 0.0f, 1.0f})
		{

			vec3 samplePos = vec3(
				floor(fx + ox) + 0.5f,
				floor(fy + oy) + 0.5f,
				floor(fz + oz) + 0.5f
			);

			float dx = (pos.x - samplePos.x);
			float dy = (pos.y - samplePos.y);
			float dz = (pos.z - samplePos.z);
			float ll = (dx * dx + dy * dy + dz * dz);
			float w = 0.0f;

			float l = sqrt(ll);

			if(ll < 1.0f){
				// exponential filter
				// w = __expf(-ll * 0.5f);
				// w = clamp(w, 0.0f, 1.0f);
				
				// linear filter
				w = 1.0 - l;
			}else{
				w = 0.0;
			}

			if(w > 0.0f){

				uint64_t W = clamp(100.0f * w, 1.0f, 100.0f);

				uint32_t ix = clamp(samplePos.x, 0.0f, fGridSize - 1.0f);
				uint32_t iy = clamp(samplePos.y, 0.0f, fGridSize - 1.0f);
				uint32_t iz = clamp(samplePos.z, 0.0f, fGridSize - 1.0f);

				uint32_t voxelIndex = ix + gridSize * iy + gridSize * gridSize * iz;

				bool isCenter = ox == 0.0f && oy == 0.0f && oz == 0.0f;
				bool isNeighbor = !isCenter;
				if(isNeighbor){
					uint64_t* cell = (uint64_t*)&voxelGrid[4 * voxelIndex + 0];

					uint32_t currentW = voxelGrid[4 * voxelIndex + 3];

					if(currentW > 0){
						uint8_t* rgba = (uint8_t*)&point.color;
						uint64_t R = W * rgba[0];
						uint64_t G = W * rgba[1];
						uint64_t B = W * rgba[2];
						atomicAdd(cell + 0, uint64_t(R | (G << 32)));
						atomicAdd(cell + 1, uint64_t(B | (W << 32)));
					}
				}

				
			}
		}
	}
}

void main_voxelize(
	Allocator& allocator, 
	Box3 box, 
	int numPoints,
	void* nnnodes,
	uint32_t numNodes,
	void* sssorted)
{
	// Point* sorted = (Point*)sssorted;
	Node* nodes = (Node*)nnnodes;

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	NodePtr* workload              =  allocator.alloc<NodePtr*>(sizeof(Node) * numNodes, "workload");
	uint32_t& workloadSize         = *allocator.alloc<uint32_t*>(sizeof(uint32_t), "workload counter");
	uint32_t& totalVoxelBufferSize = *allocator.alloc<uint32_t*>(4, "total voxel buffer counter");
	uint32_t& nodeVoxelBufferSize  = *allocator.alloc<uint32_t*>(4, "node voxel buffer counter");
	uint32_t& clearCounter         = *allocator.alloc<uint32_t*>(4, "clear counter");
	uint32_t& nodeCounter          = *allocator.alloc<uint32_t*>(4);

	if(isFirstThread()){
		totalVoxelBufferSize = 0;
		nodeVoxelBufferSize = 0;
		clearCounter = 0;
		nodeCounter = 0;
	}

	grid.sync();
	
	int gridSize               = VOXEL_GRID_SIZE;
	// float fGridSize            = gridSize;
	int numCells               = gridSize * gridSize * gridSize;
	int acceptedCapacity       = 200'000;
	uint32_t& workIndexCounter = *allocator.alloc<uint32_t*>(4, "work index counter");
	uint64_t acceptedByteSize  = sizeof(uint32_t) * acceptedCapacity;
	uint32_t* accepteds           = allocator.alloc<uint32_t*>(grid.num_blocks() * acceptedByteSize, "list of accepted indices");
	uint32_t* accepted            = accepteds + grid.block_rank() * acceptedCapacity;

	// Create one voxelgrid per workgroup, and a <voxelGrid> pointer that points to the active workgroup's memory
	uint64_t voxelGridByteSize = 4 * sizeof(uint32_t) * numCells;
	uint32_t* voxelGrids       = allocator.alloc<uint32_t*>(grid.num_blocks() * voxelGridByteSize, "voxel sampling grids");
	uint32_t* voxelGrid        = voxelGrids + grid.block_rank() * 4 * numCells;

	allocator.alloc<uint32_t*>(10000 * 16);
	uint64_t& globalAllocatorOffset = *allocator.alloc<uint64_t*>(8);
	allocator.alloc<uint32_t*>(10000 * 16);

	__shared__ uint32_t sh_workIndex;
	__shared__ uint32_t sh_numAccepted;
	__shared__ uint32_t sh_clearGrid[clearGridSize * clearGridSize * clearGridSize];

	// initially clear all voxel grids
	clearBuffer(voxelGrids, 0, grid.num_blocks() * voxelGridByteSize, 0);

	grid.sync();

	if(isFirstThread()){
		globalAllocatorOffset = allocator.offset;

		// printf("allocator.offset: ");
		// printNumber(allocator.offset, 10);
		// printf("\n");
	}
	grid.sync();

	// loop from bottom of hierarchy to top until all work done, 
	// but limit loop range to max octree depth to be safe
	for(int abc = 0; abc < 20; abc++){

		grid.sync();

		computeWorkload(nodes, numNodes, workload, workloadSize);
		if(grid.thread_rank() == 0){
			workIndexCounter = 0;
		}
		grid.sync();

		if(workloadSize == 0) break;

		while(workIndexCounter < workloadSize){

			block.sync();

			if(block.thread_rank() == 0){
				sh_workIndex = atomicAdd(&workIndexCounter, 1);
				sh_numAccepted = 0;
			}

			block.sync();

			if(sh_workIndex >= workloadSize) break;

			// retrieve the node that this block should process
			Node* node = workload[sh_workIndex];
			vec3 boxSize = node->max - node->min;
			vec3 childSize = boxSize * 0.5f;

			block.sync();

			{ // this assumes a workgroup size of 256!!!
				if(block.num_threads() != 256)
				if(block.thread_rank() == 0)
				{
					printf("error, expecting a workgroup size of 256");
				}

				sh_clearGrid[2 * block.thread_rank() + 0] = 0;
				sh_clearGrid[2 * block.thread_rank() + 1] = 0;
			}

			block.sync();

			// first, central projection
			for(int childIndex = 0; childIndex < 8; childIndex++){
				Node* child = node->children[childIndex];

				if(child == nullptr) continue;

				block.sync();

				// POINTS
				voxelizePrimitives_central(
					child->points, child->numPoints, node, 
					gridSize, voxelGrid, boxSize, sh_numAccepted, accepted, sh_clearGrid);

				block.sync();

				// VOXELS
				voxelizePrimitives_central(
					child->voxels, child->numVoxels, node, 
					gridSize, voxelGrid, boxSize, sh_numAccepted, accepted, sh_clearGrid);

				block.sync();
			}

			block.sync();

			// then, neighbor projection.
			// neighbors only modify cells that were
			// occupied by the central projection,
			// which allows us to quickly clear only relevant cells
			for(int childIndex = 0; childIndex < 8; childIndex++){
				Node* child = node->children[childIndex];

				if(child == nullptr) continue;

				block.sync();

				// POINTS
				voxelizePrimitives_neighbors(
					child->points, child->numPoints, node, 
					gridSize, voxelGrid, boxSize, sh_numAccepted, accepted, sh_clearGrid);

				block.sync();

				// VOXELS
				voxelizePrimitives_neighbors(
					child->voxels, child->numVoxels, node, 
					gridSize, voxelGrid, boxSize, sh_numAccepted, accepted, sh_clearGrid);

				block.sync();
			}

			block.sync();

			// now allocate memory for the voxels of this node
			Point* voxelBuffer = nullptr;
			if(block.thread_rank() == 0){
				uint64_t bufferOffset = atomicAdd(&globalAllocatorOffset, 16ull * sh_numAccepted);
				voxelBuffer = reinterpret_cast<Point*>(allocator.buffer + bufferOffset);
				node->voxels = voxelBuffer;
				node->numVoxels = sh_numAccepted;
			}

			// { // DEBUG: clear voxel buffer first (should not be necessary)
			// 	block.sync();

			// 	int numIterations = sh_numAccepted / block.num_threads() + 1;
			// 	for(int it = 0; it <= numIterations; it++){
			// 		int index = block.num_threads() * it + block.thread_rank();

			// 		if(index >= sh_numAccepted) continue;

			// 		Point point;
			// 		point.x = 0;
			// 		point.y = 0;
			// 		point.z = 0;
			// 		point.color = 0;
			// 		node->voxels[index] = point;

			// 	}
			// }

			block.sync();

			// EXTRACT
			int numIterations = sh_numAccepted / block.num_threads() + 1;
			for(int it = 0; it < numIterations; it++){
				int index = block.num_threads() * it + block.thread_rank();

				if(index >= sh_numAccepted) continue;

				uint32_t voxelIndex = accepted[index];

				uint32_t R = voxelGrid[4 * voxelIndex + 0];
				uint32_t G = voxelGrid[4 * voxelIndex + 1];
				uint32_t B = voxelGrid[4 * voxelIndex + 2];
				uint32_t W = voxelGrid[4 * voxelIndex + 3];

				// bool occupied = (W & (1u << 31u)) != 0u;
				// W = W & 0b0111111'11111111'11111111'11111111;

				uint32_t color;
				uint8_t* rgba = (uint8_t*)&color;
				rgba[0] = R / W;
				rgba[1] = G / W;
				rgba[2] = B / W;

				int ix = voxelIndex % gridSize;
				int iy = (voxelIndex % (gridSize * gridSize)) / gridSize;
				int iz = voxelIndex / (gridSize * gridSize);

				float x = (float(ix) + 0.5f) * boxSize.x / float(gridSize);
				float y = (float(iy) + 0.5f) * boxSize.y / float(gridSize);
				float z = (float(iz) + 0.5f) * boxSize.z / float(gridSize);
				// float cubeSize = boxSize.x / float(gridSize);

				vec3 pos = {x, y, z};
				pos = pos + node->min;

				Point voxel;
				voxel.x = pos.x;
				voxel.y = pos.y;
				voxel.z = pos.z;
				voxel.color = color;

				bool outsideX = pos.x < node->min.x || pos.x >= node->max.x;
				bool outsideY = pos.y < node->min.y || pos.y >= node->max.y;
				bool outsideZ = pos.z < node->min.z || pos.z >= node->max.z;
				if(outsideX || outsideY || outsideZ){
					// printf("out of bounds \n");
				}

				// if(node->level == 3 && node->voxelIndex == 99){
				// 	if(voxel.y > 2.32621408f - 0.01f && voxel.y < 2.32621408f + 0.01f)
				// 	if(voxel.z > 0.841976047f - 0.01f && voxel.z < 0.841976047f + 0.01f)
				// 	// if(index < 10)
				// 	{
				// 		printf("xyz: %f, %f, %f \n", voxel.x, voxel.y, voxel.z);
				// 	}
				// }

				// if(voxel.x == 0.0f && voxel.y == 0.0f && voxel.z == 0.0f){
				// 	printf("xyz: %f, %f, %f \n", voxel.x, voxel.y, voxel.z);
				// }
				

				node->voxels[index] = voxel;

				// Since only cells that contain points are affected,
				// we can directly clear the cell now.
				// (neighbors dont modify cells without actual geometry)
				voxelGrid[4 * voxelIndex + 0] = 0;
				voxelGrid[4 * voxelIndex + 1] = 0;
				voxelGrid[4 * voxelIndex + 2] = 0;
				voxelGrid[4 * voxelIndex + 3] = 0;
			}

			if(block.thread_rank() == 0){
				node->dbg = atomicAdd(&nodeCounter, 1);
			}
			
			block.sync();
		}
	}

	grid.sync();

	// update allocator.offset for subsequent allocations in "kernel.cu"
	allocator.offset = globalAllocatorOffset;

	// PRINT("smallVolumeNodeCounter: %i \n", smallVolumeNodeCounter);
	// PRINT("smallVolumePointCounter: %i k \n", (smallVolumePointCounter / 1000) );
}

};