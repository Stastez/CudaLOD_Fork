#include "hip/hip_runtime.h"


namespace singlecell{

#include <hip/hip_cooperative_groups.h>
#include "lib.h.cu"
#include "methods_common.h.cu"

namespace cg = cooperative_groups;

void voxelizePrimitives(Node* node, vec3 boxSize, uint32_t num, 
	Point* primitives, uint32_t offset, uint32_t gridSize, float fGridSize, 
	uint32_t* voxelGrid, uint32_t* voxelBufferSize, Point* voxelBuffer,
	int dbg = 0, int dbg2 = 0
){

	processRange(0, num, [&](int pointIndex){

		Point point = primitives[offset + pointIndex];

		float fx = fGridSize * (point.x - node->min.x) / boxSize.x;
		float fy = fGridSize * (point.y - node->min.y) / boxSize.y;
		float fz = fGridSize * (point.z - node->min.z) / boxSize.z;

		{ // AVERAGE


			uint32_t ix = clamp(fx, 0.0f, fGridSize - 1.0f);
			uint32_t iy = clamp(fy, 0.0f, fGridSize - 1.0f);
			uint32_t iz = clamp(fz, 0.0f, fGridSize - 1.0f);

			uint32_t voxelIndex = ix + gridSize * iy + gridSize * gridSize * iz;

			uint32_t R = ((point.color >>  0) & 0xff);
			uint32_t G = ((point.color >>  8) & 0xff);
			uint32_t B = ((point.color >> 16) & 0xff);
			int A = 1;

			uint64_t* voxelGrid64 = (uint64_t*)voxelGrid;

			uint64_t RG = (uint64_t(G) << 32ull) | R;
			uint64_t BA = (uint64_t(A) << 32ull) | B;
			atomicAdd(&voxelGrid64[2 * voxelIndex + 0], RG);
			auto old = atomicAdd(&voxelGrid64[2 * voxelIndex + 1], BA);

			if(old == 0){
				uint32_t rindex = atomicAdd(voxelBufferSize, 1);
				voxelBuffer[rindex].color = voxelIndex;
			}
		}


		// { // WEIGHTED AVERAGE
		// 	vec3 pos = {fx, fy, fz};
		// 	vec3 samplePos = vec3(
		// 		floor(fx) + 0.5f,
		// 		floor(fy) + 0.5f,
		// 		floor(fz) + 0.5f
		// 	);

		// 	float dx = (pos.x - samplePos.x);
		// 	float dy = (pos.y - samplePos.y);
		// 	float dz = (pos.z - samplePos.z);
		// 	float ll = (dx * dx + dy * dy + dz * dz);
		// 	float w = 0.0f;

		// 	if(ll < 1.0f){
		// 		w = __expf(-ll * 3.5f);
		// 		w = clamp(w, 0.0f, 1.0f);
		// 	}

		// 	if(w > 0.0f)
		// 	{

		// 		int W = clamp(100.0f * w, 1.0f, 100.0f);
		// 		// int W = 1;

		// 		uint32_t ix = clamp(samplePos.x, 0.0f, fGridSize - 1.0f);
		// 		uint32_t iy = clamp(samplePos.y, 0.0f, fGridSize - 1.0f);
		// 		uint32_t iz = clamp(samplePos.z, 0.0f, fGridSize - 1.0f);

		// 		uint32_t voxelIndex = ix + gridSize * iy + gridSize * gridSize * iz;

		// 		uint32_t R = W * ((point.color >>  0) & 0xff);
		// 		uint32_t G = W * ((point.color >>  8) & 0xff);
		// 		uint32_t B = W * ((point.color >> 16) & 0xff);

		// 		atomicAdd((uint64_t*)&voxelGrid[4 * voxelIndex + 0], (((uint64_t)G) << 32) | R);
		// 		auto old = atomicAdd((uint64_t*)&voxelGrid[4 * voxelIndex + 2], (((uint64_t)W) << 32) | B);

		// 		if(old == 0){
		// 			uint32_t rindex = atomicAdd(voxelBufferSize, 1);
		// 			voxelBuffer[rindex].color = voxelIndex;
		// 		}
		// 	}
		// }
		
	});
}

void main_voxelize(
	Allocator& allocator, 
	Box3 box, 
	int numPoints,
	void* nnnodes,
	uint32_t numNodes,
	void* sssorted)
{
	// Point* sorted = (Point*)sssorted;
	Node* nodes = (Node*)nnnodes;

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	NodePtr* workload              =  allocator.alloc<NodePtr*>(sizeof(Node) * numNodes, "workload");
	uint32_t& workloadSize         = *allocator.alloc<uint32_t*>(sizeof(uint32_t), "workload counter");
	uint32_t& totalVoxelBufferSize = *allocator.alloc<uint32_t*>(4, "total voxel buffer counter");
	uint32_t& nodeVoxelBufferSize  = *allocator.alloc<uint32_t*>(4, "node voxel buffer counter");
	uint32_t& clearCounter         = *allocator.alloc<uint32_t*>(4, "clear counter");

	if(isFirstThread()){
		totalVoxelBufferSize = 0;
		nodeVoxelBufferSize = 0;
		clearCounter = 0;
	}

	grid.sync();

	{
		int gridSize        = VOXEL_GRID_SIZE;
		float fGridSize     = gridSize;
		int numCells        = gridSize * gridSize * gridSize;
		uint32_t* voxelGrid = allocator.alloc<uint32_t*>(sizeof(uint32_t) * 4 * numCells, "voxel sampling grid RG");
		// uint32_t* voxelGrid_RG = allocator.alloc<uint32_t*>(sizeof(uint32_t) * 2 * numCells, "voxel sampling grid RG");
		// uint32_t* voxelGrid_BA = allocator.alloc<uint32_t*>(sizeof(uint32_t) * 2 * numCells, "voxel sampling grid BA");

		clearBuffer(voxelGrid, 0, sizeof(uint32_t) * 4 * numCells, 0);
		// clearBuffer(voxelGrid_RG, 0, sizeof(uint32_t) * 2 * numCells, 0);
		// clearBuffer(voxelGrid_BA, 0, sizeof(uint32_t) * 2 * numCells, 0);

		for(int abc = 0; abc < 12; abc++){

			grid.sync();

			auto t_create_workload = Timer::start("create workload");
			{ // FIND DEEPEST EMPTY INNER NODES (TO FILL WITH VOXELS)
				if(isFirstThread()){ 
					workloadSize = 0;
				}
				grid.sync();

				processRange(0, numNodes, [&](int nodeIndex){
					Node* node = &nodes[nodeIndex];
					
					int numSamplesInChildren = 0;
					bool allChildrenNonempty = true;
					for(int childIndex = 0; childIndex < 8; childIndex++){
						Node* child = node->children[childIndex];

						if(child){
							numSamplesInChildren += child->numPoints + child->numVoxels;
							if((child->numPoints + child->numVoxels) == 0){
								allChildrenNonempty = false;
							}
						}
					}
				
					bool isEmpty = node->numPoints == 0 && node->numVoxels == 0;

					if(isEmpty && allChildrenNonempty){
						uint32_t targetIndex = atomicAdd(&workloadSize, 1);
						workload[targetIndex] = node;
					}
				});

				grid.sync();
			}
			t_create_workload.stop();


			if(workloadSize == 0){
				// we're done!
				break;
			}

			{ // CREATE VOXEL LOD FOR DEEPEST EMPTY INNER NODES

				for(int nodeIndex = 0; nodeIndex < workloadSize; nodeIndex++){
					
					Node* node = workload[nodeIndex];
					Point* voxelBuffer = reinterpret_cast<Point*>(allocator.buffer + allocator.offset + sizeof(Point) * totalVoxelBufferSize);

					grid.sync();

					vec3 boxSize = node->max - node->min;
					// clearBuffer(voxelGrid, 0, sizeof(uint32_t) * 4 * numCells, 0);
					// clearBuffer(voxelGrid_RG, 0, sizeof(uint32_t) * 2 * numCells, 0);
					// clearBuffer(voxelGrid_BA, 0, sizeof(uint32_t) * 2 * numCells, 0);
					// if(grid.thread_rank() == 0){
					// 	atomicAdd(&clearCounter, 1);
					// }

					grid.sync();

					// auto t_voxelize = Timer::start("voxelize");
					for(int childIndex = 0; childIndex < 8; childIndex++){
						Node* child = node->children[childIndex];
						if(child == nullptr) continue;

						grid.sync();

						// VOXELIZE POINTS
						voxelizePrimitives(node, boxSize, child->numPoints, 
							child->points, 0, gridSize, fGridSize, 
							voxelGrid, &nodeVoxelBufferSize, voxelBuffer, 0);
						
						if(abc == 0) continue;

						grid.sync();

						// VOXELIZE VOXELS
						voxelizePrimitives(node, boxSize, child->numVoxels, 
							child->voxels, 0, gridSize, fGridSize,
							voxelGrid, &nodeVoxelBufferSize, voxelBuffer,
							abc, nodeIndex
						);
					}
					// t_voxelize.stop();

					grid.sync();

					// EXTRACT VOXELS
					// auto t_extract = Timer::start("extract");
					processRange(0, nodeVoxelBufferSize, [&](int rindex){
						uint32_t index = voxelBuffer[rindex].color;
						int ix = index % gridSize;
						int iy = (index % (gridSize * gridSize)) / gridSize;
						int iz = index / (gridSize * gridSize);

						uint32_t W = voxelGrid[4 * index + 3];

						if(W > 0){
							float x = (float(ix) + 0.5f) * boxSize.x / float(gridSize);
							float y = (float(iy) + 0.5f) * boxSize.y / float(gridSize);
							float z = (float(iz) + 0.5f) * boxSize.z / float(gridSize);
							float cubeSize = boxSize.x / float(gridSize);

							vec3 pos = {x, y, z};
							pos = pos + node->min;
							vec3 size = {cubeSize, cubeSize, cubeSize};
							uint32_t color = 0x000000ff;

							uint32_t R = float(voxelGrid[4 * index + 0]) / float(W);
							uint32_t G = float(voxelGrid[4 * index + 1]) / float(W);
							uint32_t B = float(voxelGrid[4 * index + 2]) / float(W);

							R = R & 0xff;
							G = G & 0xff;
							B = B & 0xff;

							int isize = 300.0f * size.x;
							color = R | (G << 8) | (B << 16) | (isize << 24);
							// color = 0x000000ff;

							Point voxel;
							voxel.x = pos.x;
							voxel.y = pos.y;
							voxel.z = pos.z;
							voxel.color = color;

							voxelBuffer[rindex] = voxel;

							// since the singlecell method only projects to cells with actual geometry,
							// we can clear the voxelGrid directly here during extraction
							voxelGrid[4 * index + 0] = 0;
							voxelGrid[4 * index + 1] = 0;
							voxelGrid[4 * index + 2] = 0;
							voxelGrid[4 * index + 3] = 0;
						}
					});
					// t_extract.stop();

					grid.sync();


					grid.sync();
					
					if(isFirstThread()){
						node->voxels = voxelBuffer;
						node->numVoxels = nodeVoxelBufferSize;

						atomicAdd(&totalVoxelBufferSize, nodeVoxelBufferSize);
						atomicExch(&nodeVoxelBufferSize, 0);
					}
					
					grid.sync();
				}
			}
		}

		grid.sync();

		// post-allocation of generated voxels
		// we dynamically placed voxels starting from previous allocator.offset 
		// now that we've done that, we know how much memory we needed and update the allocator
		allocator.alloc<Point*>(sizeof(Point) * totalVoxelBufferSize, "all voxels");
	}

	grid.sync();

	

}

};